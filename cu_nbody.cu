#include "hip/hip_runtime.h"
/*
 *  This is a commented version of the original N-Body
 *  program, some of the internal variable has a different
 *  but better readable name. The structure and principle
 *  of this program has no big change.
 *
 *  Commented By Daniel Zhang <danielzhang0212@gmail.com>
 */


#include <stdio.h>
#include <cmath>
#include <cassert>
#include <cutil.h>
#include <omp.h>
#include "cuda_pointer.h"


/*
 * define some constants, like 
 * the number of thread of a block,
 * j blocks, and i blocks
 */
#define NTHREAD 64 // 64, 96, 128 or 192
#define NJBLOCK 16 // 8800GTS/512 has 16
#define NIBLOCK 16 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 1024
#define NBMAX 128 // NNB per block

/*
 *  user define vector
 */
template <class T>
struct myvector{
	int num;
	T *val;
	myvector(){
		num = 0;
		val = NULL;
	}
	~myvector(){
		delete [] val;
	}
	void clear(){
		num = 0;
	}
	void reserve(size_t count){
		val = new T[count];
	}
	void free(){
		delete [] val;
	}
	void push_back(const T &t){
		val[num++] = t;
	}
	size_t size(){
		return num;
	}
	T &operator[](int i){
		return val[i];
	}
};


/* 
 * define the time related functions
 * in order to record the calculation time.
 */
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}

static double time_send, time_grav;
static long long numInter


/*
 * define the structure of particle J
 * contain: position, velocity, mass and pad
 */
struct Jparticle{
	float3 pos;		// position
	float3 vel;		// velocity
	float  mass;	// mass
	float  pad;		// TODO:don't know!
	Jparticle() {}
	Jparticle(double mj, double posj[3], double velj[3]){
		pos.x = posj[0];
		pos.y = posj[1];
		pos.z = posj[2];
		mass  = mj;
		vel.x = velj[0];
		vel.y = velj[1];
		vel.z = velj[2];
	}
};


/*
 *  define the structure of particle I
 *  contain: position, velocity, pad, and h2
 */
struct Iparticle{
	float3 pos;		// position
	float3 vel;		// velocity
	float  pad;		// TODO:don't know
	float  h2;		// Threshold to dicide a neighbor
	Iparticle() {}
	Iparticle(double h2i, double posi[3], double veli[3]){
		pos.x = posi[0];
		pos.y = posi[1];
		pos.z = posi[2];
		h2    = h2i;
		vel.x = veli[0];
		vel.y = veli[1];
		vel.z = veli[2];
	}
};

/*
 * define the structure of the force
 * between two particles, including jerk
 * force, acceleration, potential, and neighborhood
 */
struct Force{
	float3 acc;		// acceleration
	float3 jrk;		// jerk force
	float  pot;		// potential
	int    nnb;     // number of neighbor particles ONE BYTE
	unsigned short  neib[NBMAX]; // 24 words
	__device__  Force(){	// created inside a kernel
		acc.x = acc.y = acc.z = 0.f;
		jrk.x = jrk.y = jrk.z = 0.f;
		pot = 0.f;
		nnb = 0;
	}
};


/*
 * calculate the FORCE contents
 * between two particles.
 * using the position and velocity of
 * two particles.
 */
__device__ void force_kernel(
				const int id, 
				const Iparticle &ip,
				const Jparticle &jp,
				Force &force)
{
	float dx = jp.pos.x - ip.pos.x;		// position diff of x
	float dy = jp.pos.y - ip.pos.y;		// position diff of y
	float dz = jp.pos.z - ip.pos.z;		// position diff of z
	float dvx = jp.vel.x - ip.vel.x;	// velocity diff of x
	float dvy = jp.vel.y - ip.vel.y;	// velocity diff of y
	float dvz = jp.vel.z - ip.vel.z;	// velocity diff of z

	float dist_square = dx*dx + dy*dy + dz*dz;

	float inverse = rsqurtf(dist_square);	// used as denominator

	if(dist_square < ip.h2)	// if the distance is smaller than a value
	{
		// add this particle into neibour list
		force.neib[force.nnb & (NBMAX-1)] = (unsigned)id;	// do modulus
		/* NBMAX-1=(11111111) in binary */
		force.nnb++;

		/* neighbor's force does not count */
		inverse = 0.f;
	}
	
	float inverse_square = inverse * inverse;
	float potential = jp.mass * inverse;		// potential
	float acc_para = mass_inverse * inverse;	// accelerate para
	
	rv = -3.f * inverse_square * (dx*dvx + dy*dvy + dz*dvz);

	/* the final calculation of the FORCE */
#ifdef POTENTIAL
	force.pot += potential;
#endif
	force.acc.x += acc_para * dx;
	force.acc.y += acc_para * dy;
	force.acc.z += acc_para * dz;
	force.jrk.x += acc_para * (dvx + rv * dx);	
	force.jrk.y += acc_para * (dvy + rv * dy);	
	force.jrk.z += acc_para * (dvz + rv * dz);	

	// FUNCTION DONE
}

/*
 *  Main kernel of the n-body forces calculation
 */
__global__ void KERNEL(
				int nbody,	// number of bodies
				Iparticle ips[],
				Jparticle jps[],
				Force     forces[][NJBLOCK])
{
	int i_bid = blockIdx.x;
	int j_bid = blockIdx.y;
	int tid = threadIdx.x;
	int i_addr = tid + NTHREAD * i_bid;
	int j_start=(nbody/NJBLOCK) * j_bid;
	int j_end=(nbody/NJBLOCK) * j_bid +1 ;


	Iparticle ip = ips[i_addr];		// fetch the i particle
	Force fo;					

	/*
	 * loop every block of j particle,
	 * where variable j stand for the first
	 * element of the Jblock
	 */
	for(int j=j_start;j<j_end;j+=NTHREAD)
	{
		__shared__ Jparticle jpshare[NTHREAD];	// use shared memory
		__syncthreads();

		/* copy data into shared memory */
		float4 *src = (float4 *) &jps[j];
		float4 *dst = (float4 *) jpshare;
		dst[        tid] = src[        tid];
/*          try to take this away - superfluous      */
             /* dst[NTHREAD+tid] = src[NTHREAD+tid];   */
		/*  TODO why copy 2 blocks? 
		 */
		__syncthreads();

		/* 
		 *  calculate the I particle's force, 
		 *  interactive with every J particle 
		 *  in the whole system.
		 */
		if(j_end-j<NTHREAD)
		{
			for(int partId=0;partId<jend-j;partId++)
			{
				Jparticle jp = jpshare[partId];
				force_kernel(partId, ip, jp, fo);
			}
		}
		else
		{
#pragma unroll
			for(int partId=0;partId<NTHREAD;partId++)
			{
				Jparticle jp = jpshare[partId];
				force_kernel(partId, ip, jp, fo);
			}
		}
	}
	/*
	 *  the fo's calculation is complete,
	 *  save it into buffer. fo stand for 
	 *  the i's Iparticle interactive with the
	 *  j_bid's block of Jparticle
	 */
	forces[i_addr][j_bid] = fo;


}

/*
 *  The functions below are CPU functions
 */

static cudaPointer <Jparticle> jps;
static cudaPointer <Iparticle> ips;
static cudaPointer <Force[NJBLOCK]> forces;

#define MAX_CPU 1
static myvector<int> nblist[MAX_CPU];
static int nbody, nbodymax;

/*
 *  the initial and end functions
 */
void GPUNB_open(int nbmax){
	time_send = time_grav = 0.0;
	numInter = 0;

	jpbuf.allocate(nbmax + NTHREAD);
	ipbuf.allocate(NIMAX);
	fobuf.allocate(NIMAX);
	nbodymax = nbmax;
#pragma omp parallel
	{
                int tid = 0;
		nblist[tid].reserve(nbmax);
	}
}
void GPUNB_close(){
	jpbuf.free();
	ipbuf.free();
	fobuf.free();
	nbodymax = 0;

#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}
void GPUNB_send(
		int nj,
		double mj[],
		double xj[][3],
		double vj[][3]){
	time_send -= get_wtime();
	nbody = nj;
	assert(nbody <= nbodymax);
	for(int j=0; j<nj; j++){
		jpbuf[j] = Jparticle(mj[j], xj[j], vj[j]);
	}

	jpbuf.htod(nj);
	time_send += get_wtime();
}



/*
 *  MAIN FUNCTION of this program.
 *  Call the kernel.
 */
void GPUNB_regf(
				int nI,			// number of I particle
				double h2[], 	// one para of Iparticle
				double xi[][3], // position vector
				double vi[][3], // velocity vector
				double acc[][3],// acceleration vecotor
				double jrk[][3],// force vector
				double pot[],	// potential
				int lmax, 		// TODO
				int nbMax,		// max number of neighbor
				int *listbase)  // TODO
{

	/* get initial time */
	time_grav -= get_wtime();
	numInter += ni * nbody;
	assert(0 < ni && ni <= NIMAX)

	for(int i=0;i<nI; i++)
	{
		ips[i] = Iparticle(h2[i],xi[i],vi[i]);
	}

	/* load the particles into device memory */
	ipbuf.htod(ni);	

	/* start the kernel */
	int niblock = 1 + (nI - 1) / NTHREAD;
	dim3 grid(niblock , NJBLOCK, 1);
	dim3 threads(NTHREAD, 1,1,);
	KERNEL <<< grid, threads >>> (nbody,ips,jps,forces)

	forces.dtoh(nI);	
	// reduction phase

#pragma omp parallel for
	for(int i=0;i<nI;i++)
	{
		// TODO I think "tid" should be "i"
		//      For it has never changed in 
		//      every loop.
		int tid=0;	
		double ax=0,ay=0,az=0;	// acceleration
		double jx=0,jy=0,jz=0;	// jerk force
#ifdef POTENTIAL
		double poti=0;
#endif
		for(int jblock=0;jblock<NJBLOCK;jblock++)
		{
			Force &fo = forces[i][jblock];
			ax += fo.acc.x;
			ay += fo.acc.y;
			az += fo.acc.z;
			jx += fo.jrk.x;
			jy += fo.jrk.y;
			jz += fo.jrk.z;
#ifdef POTENTIAL
			poti += fo.pot;
#endif
		}
		/* save the sum of individual result */
		acc[i][0] = ax;
		acc[i][1] = ay;
		acc[i][2] = az;
		jrk[i][0] = jx;
		jrk[i][1] = jy;
		jrk[i][2] = jz;
#ifdef POTENTIAL
		pot[i] = poti;
#endif

		/* TODO something about neighbors */
		bool overflow = false;
		for(int jblock=0;jblock<NJBLOCK;jblock++)
		{
			Force &fo = forces[i][jblock];
			int jstart = (nbody / NJBLOCK) * jblock;
			if(fo.nnb <= NBMAX)
			{
				for(int k=0;k<f.nnb;k++)
				{
					int nb = fo.neib[k];
					while(nb<jstart) 
/*                  increment should be determined by size of block  ?? */
					nb += (1<<16);
					nblist[tid].push_back(nb);
				}
			}
			else
			{
				overflow = true;
			}
		}

		int * nnbp = listbase + lmax * i;	// number of neighbor particles
		int * nblistp = nnbp +1;
		int nnb = nblist[tid].size();
		if(nnb > nbmax) 
			overflow =true;

		if(overflow)
		{
			*nnbp = -1;
		}
		else
		{
			*nnbp = nnb;
			for(int k=0;k<nnb;k++)
			{
				nblistp[k]=nblist[tid][k];
			}
		}
			
		

	}

}

/* TODO combine programming between C and C++ */
extern "C" {
	void gpunb_open_(int *nbmax){
		GPUNB_open(*nbmax);
	}
	void gpunb_close_(){
		GPUNB_close();
	}
	void gpunb_send_(
			int *nj,
			double mj[],
			double xj[][3],
			double vj[][3]){
		GPUNB_send(*nj, mj, xj, vj);
	}
	void gpunb_regf_(
			int *ni,
			double h2[],
			double xi[][3],
			double vi[][3],
			double acc[][3],
			double jrk[][3],
			double pot[],
			int *lmax,
			int *nbmax,
			int *list){ // list[][lmax]
		GPUNB_regf(*ni, h2, xi, vi, acc, jrk, pot, *lmax, *nbmax, list);
	}
}

